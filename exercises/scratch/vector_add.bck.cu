#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void add(float *a, float *b, float *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    c[index] = a[index] + b[index];
  }
}

void fill_random(float *buf, size_t n) {
  for (auto i = 0; i < n; i++){
    buf[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
  }
}


int main(int argc, char **argv) {
  srand(static_cast <unsigned> (time(0)));
  int n = atoi(argv[1]);
  float *a, *b, *c;
  float *d_a, *d_b, *d_c;
  size_t dsize = n * sizeof(float);

  a = new float[dsize];
  b = new float[dsize];
  c = new float[dsize];

  fill_random(a, n);
  fill_random(b, n);

  hipMalloc(&d_a, dsize);
  hipMalloc(&d_b, dsize);
  hipMalloc(&d_c, dsize);

  hipMemcpy(d_a, a, dsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, dsize, hipMemcpyHostToDevice);

  add<<<n, 1>>>(d_a, d_b, d_c, n);
  hipMemcpy(c, d_c, dsize, hipMemcpyDeviceToHost);
  printf("%f + %f = %f\n", a[0], b[0], c[0]);

}






